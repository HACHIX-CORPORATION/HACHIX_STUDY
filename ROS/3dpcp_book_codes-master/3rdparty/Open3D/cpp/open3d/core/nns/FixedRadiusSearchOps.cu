// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------
//

#include "open3d/core/Tensor.h"
#include "open3d/core/nns/FixedRadiusIndex.h"
#include "open3d/core/nns/FixedRadiusSearchImpl.cuh"
#include "open3d/core/nns/NeighborSearchAllocator.h"
#include "open3d/core/nns/NeighborSearchCommon.h"

namespace open3d {
namespace core {
namespace nns {

template <class T>
void BuildSpatialHashTableCUDA(const Tensor& points,
                               double radius,
                               const Tensor& points_row_splits,
                               const Tensor& hash_table_splits,
                               Tensor& hash_table_index,
                               Tensor& hash_table_cell_splits) {
    const hipStream_t stream = 0;
    int texture_alignment = 512;

    void* temp_ptr = nullptr;
    size_t temp_size = 0;

    open3d::core::nns::impl::BuildSpatialHashTableCUDA(
            stream, temp_ptr, temp_size, texture_alignment,
            points.GetShape()[0], points.GetDataPtr<T>(), T(radius),
            points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>());

    Device device = points.GetDevice();
    Tensor temp_tensor =
            Tensor::Empty({int64_t(temp_size)}, Dtype::UInt8, device);
    temp_ptr = temp_tensor.GetDataPtr();

    open3d::core::nns::impl::BuildSpatialHashTableCUDA(
            stream, temp_ptr, temp_size, texture_alignment,
            points.GetShape()[0], points.GetDataPtr<T>(), T(radius),
            points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>());
}

template <class T>
void FixedRadiusSearchCUDA(const Tensor& points,
                           const Tensor& queries,
                           double radius,
                           const Tensor& points_row_splits,
                           const Tensor& queries_row_splits,
                           const Tensor& hash_table_splits,
                           const Tensor& hash_table_index,
                           const Tensor& hash_table_cell_splits,
                           const Metric metric,
                           const bool ignore_query_point,
                           const bool return_distances,
                           const bool sort,
                           Tensor& neighbors_index,
                           Tensor& neighbors_row_splits,
                           Tensor& neighbors_distance) {
    const hipStream_t stream = 0;
    int texture_alignment = 512;

    Device device = points.GetDevice();
    Dtype dtype = points.GetDtype();

    NeighborSearchAllocator<T> output_allocator(device);
    void* temp_ptr = nullptr;
    size_t temp_size = 0;

    open3d::core::nns::impl::FixedRadiusSearchCUDA(
            stream, temp_ptr, temp_size, texture_alignment,
            neighbors_row_splits.GetDataPtr<int64_t>(), points.GetShape()[0],
            points.GetDataPtr<T>(), queries.GetShape()[0],
            queries.GetDataPtr<T>(), T(radius), points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            queries_row_splits.GetShape()[0],
            queries_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>(), metric, ignore_query_point,
            return_distances, output_allocator);

    Tensor temp_tensor =
            Tensor::Empty({int64_t(temp_size)}, Dtype::UInt8, device);
    temp_ptr = temp_tensor.GetDataPtr();

    open3d::core::nns::impl::FixedRadiusSearchCUDA(
            stream, temp_ptr, temp_size, texture_alignment,
            neighbors_row_splits.GetDataPtr<int64_t>(), points.GetShape()[0],
            points.GetDataPtr<T>(), queries.GetShape()[0],
            queries.GetDataPtr<T>(), T(radius), points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            queries_row_splits.GetShape()[0],
            queries_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>(), metric, ignore_query_point,
            return_distances, output_allocator);

    Tensor indices_unsorted = output_allocator.NeighborsIndex();
    Tensor distances_unsorted = output_allocator.NeighborsDistance();

    if (!sort) {
        neighbors_index = indices_unsorted;
        neighbors_distance = distances_unsorted;
    } else {
        // Sort indices & distances.
        temp_ptr = nullptr;
        temp_size = 0;

        int64_t num_indices = indices_unsorted.GetShape()[0];
        int64_t num_segments = neighbors_row_splits.GetShape()[0] - 1;
        Tensor indices_sorted =
                Tensor::Empty({num_indices}, Dtype::Int32, device);
        Tensor distances_sorted = Tensor::Empty({num_indices}, dtype, device);

        // Determine temp_size for sorting
        open3d::core::nns::impl::SortPairs(
                temp_ptr, temp_size, texture_alignment, num_indices,
                num_segments, neighbors_row_splits.GetDataPtr<int64_t>(),
                indices_unsorted.GetDataPtr<int32_t>(),
                distances_unsorted.GetDataPtr<T>(),
                indices_sorted.GetDataPtr<int32_t>(),
                distances_sorted.GetDataPtr<T>());

        temp_tensor = Tensor::Empty({int64_t(temp_size)}, Dtype::UInt8, device);
        temp_ptr = temp_tensor.GetDataPtr();

        // Actually run the sorting.
        open3d::core::nns::impl::SortPairs(
                temp_ptr, temp_size, texture_alignment, num_indices,
                num_segments, neighbors_row_splits.GetDataPtr<int64_t>(),
                indices_unsorted.GetDataPtr<int32_t>(),
                distances_unsorted.GetDataPtr<T>(),
                indices_sorted.GetDataPtr<int32_t>(),
                distances_sorted.GetDataPtr<T>());
        neighbors_index = indices_sorted;
        neighbors_distance = distances_sorted;
    }
}

template <class T>
void HybridSearchCUDA(const Tensor& points,
                      const Tensor& queries,
                      double radius,
                      int max_knn,
                      const Tensor& points_row_splits,
                      const Tensor& queries_row_splits,
                      const Tensor& hash_table_splits,
                      const Tensor& hash_table_index,
                      const Tensor& hash_table_cell_splits,
                      const Metric metric,
                      Tensor& neighbors_index,
                      Tensor& neighbors_count,
                      Tensor& neighbors_distance) {
    const hipStream_t stream = 0;

    Device device = points.GetDevice();

    NeighborSearchAllocator<T> output_allocator(device);

    open3d::core::nns::impl::HybridSearchCUDA(
            stream, points.GetShape()[0], points.GetDataPtr<T>(),
            queries.GetShape()[0], queries.GetDataPtr<T>(), T(radius), max_knn,
            points_row_splits.GetShape()[0],
            points_row_splits.GetDataPtr<int64_t>(),
            queries_row_splits.GetShape()[0],
            queries_row_splits.GetDataPtr<int64_t>(),
            hash_table_splits.GetDataPtr<uint32_t>(),
            hash_table_cell_splits.GetShape()[0],
            hash_table_cell_splits.GetDataPtr<uint32_t>(),
            hash_table_index.GetDataPtr<uint32_t>(), metric, output_allocator);

    neighbors_index = output_allocator.NeighborsIndex();
    neighbors_distance = output_allocator.NeighborsDistance();
    neighbors_count = output_allocator.NeighborsCount();
}

#define INSTANTIATE_BUILD(T)                                                  \
    template void BuildSpatialHashTableCUDA<T>(                               \
            const Tensor& points, double radius,                              \
            const Tensor& points_row_splits, const Tensor& hash_table_splits, \
            Tensor& hash_table_index, Tensor& hash_table_cell_splits);

#define INSTANTIATE_RADIUS(T)                                                  \
    template void FixedRadiusSearchCUDA<T>(                                    \
            const Tensor& points, const Tensor& queries, double radius,        \
            const Tensor& points_row_splits, const Tensor& queries_row_splits, \
            const Tensor& hash_table_splits, const Tensor& hash_table_index,   \
            const Tensor& hash_table_cell_splits, const Metric metric,         \
            const bool ignore_query_point, const bool return_distances,        \
            const bool sort, Tensor& neighbors_index,                          \
            Tensor& neighbors_row_splits, Tensor& neighbors_distance);

#define INSTANTIATE_HYBRID(T)                                                  \
    template void HybridSearchCUDA<T>(                                         \
            const Tensor& points, const Tensor& queries, double radius,        \
            int max_knn, const Tensor& points_row_splits,                      \
            const Tensor& queries_row_splits, const Tensor& hash_table_splits, \
            const Tensor& hash_table_index,                                    \
            const Tensor& hash_table_cell_splits, const Metric metric,         \
            Tensor& neighbors_index, Tensor& neighbors_count,                  \
            Tensor& neighbors_distance);

INSTANTIATE_BUILD(float)
INSTANTIATE_BUILD(double)

INSTANTIATE_RADIUS(float)
INSTANTIATE_RADIUS(double)

INSTANTIATE_HYBRID(float)
INSTANTIATE_HYBRID(double)

}  // namespace nns
}  // namespace core
}  // namespace open3d
