// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------
//

#define EIGEN_USE_GPU
#include "BallQueryOpKernel.h"
#include "open3d/ml/Helper.h"
#include "open3d/ml/contrib/BallQuery.cuh"
#include "open3d/ml/contrib/cuda_utils.h"

using namespace open3d;
using namespace open3d::ml;
using namespace open3d::ml::contrib;
using namespace tensorflow;

class BallQueryOpKernelCUDA : public BallQueryOpKernel {
public:
    explicit BallQueryOpKernelCUDA(OpKernelConstruction *construction)
        : BallQueryOpKernel(construction) {}

    void Kernel(tensorflow::OpKernelContext *context,
                int b,
                int n,
                int m,
                float radius,
                int nsample,
                const float *new_xyz,
                const float *xyz,
                int *idx) {
        // dataset: (B, N, 3)
        // tmp: (B, N)
        // output:
        //      idx: (B, M)

        auto stream = context->eigen_gpu_device().stream();

        hipError_t err;

        dim3 blocks(DIVUP(m, THREADS_PER_BLOCK),
                    b);  // blockIdx.x(col), blockIdx.y(row)
        dim3 threads(THREADS_PER_BLOCK);

        ball_query_kernel<<<blocks, threads, 0, stream>>>(
                b, n, m, radius, nsample, new_xyz, xyz, idx);
        // hipDeviceSynchronize();  // for using printf in kernel function
        err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "CUDA kernel failed : %s\n",
                    hipGetErrorString(err));
            exit(-1);
        }
    }
};

REGISTER_KERNEL_BUILDER(Name("Open3DBallQuery").Device(DEVICE_GPU),
                        BallQueryOpKernelCUDA);
