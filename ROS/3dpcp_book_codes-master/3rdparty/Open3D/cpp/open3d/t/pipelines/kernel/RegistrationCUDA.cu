#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/Tensor.h"
#include "open3d/t/pipelines/kernel/Reduction6x6Impl.cuh"
#include "open3d/t/pipelines/kernel/RegistrationImpl.h"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"
#include "open3d/t/pipelines/registration/RobustKernel.h"
#include "open3d/t/pipelines/registration/RobustKernelImpl.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {

const int kThread1DUnit = 256;

template <typename scalar_t, typename func_t>
__global__ void ComputePosePointToPlaneKernelCUDA(
        const scalar_t *source_points_ptr,
        const scalar_t *target_points_ptr,
        const scalar_t *target_normals_ptr,
        const int64_t *correspondence_indices,
        const int n,
        scalar_t *global_sum,
        func_t GetWeightFromRobustKernel) {
    __shared__ scalar_t local_sum0[kThread1DUnit];
    __shared__ scalar_t local_sum1[kThread1DUnit];
    __shared__ scalar_t local_sum2[kThread1DUnit];

    const int tid = threadIdx.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (workload_idx >= n) return;

    scalar_t J_ij[6] = {0}, reduction[29] = {0};
    scalar_t r = 0;

    bool valid = GetJacobianPointToPlane<scalar_t>(
            workload_idx, source_points_ptr, target_points_ptr,
            target_normals_ptr, correspondence_indices, J_ij, r);

    scalar_t w = GetWeightFromRobustKernel(r);

    if (valid) {
        // Dump J, r into JtJ and Jtr
        int i = 0;
        for (int j = 0; j < 6; ++j) {
            for (int k = 0; k <= j; ++k) {
                reduction[i] += J_ij[j] * w * J_ij[k];
                ++i;
            }
            reduction[21 + j] += J_ij[j] * w * r;
        }
        reduction[27] += r;
        reduction[28] += 1;
    }

    ReduceSum6x6LinearSystem<scalar_t, kThread1DUnit>(tid, valid, reduction,
                                                      local_sum0, local_sum1,
                                                      local_sum2, global_sum);
}

void ComputePosePointToPlaneCUDA(const core::Tensor &source_points,
                                 const core::Tensor &target_points,
                                 const core::Tensor &target_normals,
                                 const core::Tensor &correspondence_indices,
                                 core::Tensor &pose,
                                 float &residual,
                                 int &inlier_count,
                                 const core::Dtype &dtype,
                                 const core::Device &device,
                                 const registration::RobustKernel &kernel) {
    int n = source_points.GetLength();

    core::Tensor global_sum = core::Tensor::Zeros({29}, dtype, device);
    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t *global_sum_ptr = global_sum.GetDataPtr<scalar_t>();

        DISPATCH_ROBUST_KERNEL_FUNCTION(
                kernel.type_, scalar_t, kernel.scaling_parameter_,
                kernel.shape_parameter_, [&]() {
                    ComputePosePointToPlaneKernelCUDA<<<
                            blocks, threads, 0, core::cuda::GetStream()>>>(
                            source_points.GetDataPtr<scalar_t>(),
                            target_points.GetDataPtr<scalar_t>(),
                            target_normals.GetDataPtr<scalar_t>(),
                            correspondence_indices.GetDataPtr<int64_t>(), n,
                            global_sum_ptr, GetWeightFromRobustKernel);
                });
    });

    core::cuda::Synchronize();

    DecodeAndSolve6x6(global_sum, pose, residual, inlier_count);
}

template <typename scalar_t, typename funct_t>
__global__ void ComputePoseColoredICPKernelCUDA(
        const scalar_t *source_points_ptr,
        const scalar_t *source_colors_ptr,
        const scalar_t *target_points_ptr,
        const scalar_t *target_normals_ptr,
        const scalar_t *target_colors_ptr,
        const scalar_t *target_color_gradients_ptr,
        const int64_t *correspondence_indices,
        const scalar_t sqrt_lambda_geometric,
        const scalar_t sqrt_lambda_photometric,
        const int n,
        scalar_t *global_sum,
        funct_t GetWeightFromRobustKernel) {
    __shared__ scalar_t local_sum0[kThread1DUnit];
    __shared__ scalar_t local_sum1[kThread1DUnit];
    __shared__ scalar_t local_sum2[kThread1DUnit];

    const int tid = threadIdx.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (workload_idx >= n) return;

    scalar_t J_G[6] = {0}, J_I[6] = {0}, reduction[29] = {0};
    scalar_t r_G = 0, r_I = 0;

    bool valid = GetJacobianColoredICP<scalar_t>(
            workload_idx, source_points_ptr, source_colors_ptr,
            target_points_ptr, target_normals_ptr, target_colors_ptr,
            target_color_gradients_ptr, correspondence_indices,
            sqrt_lambda_geometric, sqrt_lambda_photometric, J_G, J_I, r_G, r_I);

    scalar_t w_G = GetWeightFromRobustKernel(r_G);
    scalar_t w_I = GetWeightFromRobustKernel(r_I);

    if (valid) {
        // Dump J, r into JtJ and Jtr
        int i = 0;
        for (int j = 0; j < 6; ++j) {
            for (int k = 0; k <= j; ++k) {
                reduction[i] += J_G[j] * w_G * J_G[k] + J_I[j] * w_I * J_I[k];
                ++i;
            }
            reduction[21 + j] += J_G[j] * w_G * r_G + J_I[j] * w_I * r_I;
        }
        reduction[27] += r_G * r_G + r_I * r_I;
        reduction[28] += 1;
    }

    ReduceSum6x6LinearSystem<scalar_t, kThread1DUnit>(tid, valid, reduction,
                                                      local_sum0, local_sum1,
                                                      local_sum2, global_sum);
}

void ComputePoseColoredICPCUDA(const core::Tensor &source_points,
                               const core::Tensor &source_colors,
                               const core::Tensor &target_points,
                               const core::Tensor &target_normals,
                               const core::Tensor &target_colors,
                               const core::Tensor &target_color_gradients,
                               const core::Tensor &correspondence_indices,
                               core::Tensor &pose,
                               float &residual,
                               int &inlier_count,
                               const core::Dtype &dtype,
                               const core::Device &device,
                               const registration::RobustKernel &kernel,
                               const double &lambda_geometric) {
    int n = source_points.GetLength();

    core::Tensor global_sum = core::Tensor::Zeros({29}, dtype, device);
    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t sqrt_lambda_geometric =
                static_cast<scalar_t>(sqrt(lambda_geometric));
        scalar_t sqrt_lambda_photometric =
                static_cast<scalar_t>(sqrt(1.0 - lambda_geometric));

        DISPATCH_ROBUST_KERNEL_FUNCTION(
                kernel.type_, scalar_t, kernel.scaling_parameter_,
                kernel.shape_parameter_, [&]() {
                    ComputePoseColoredICPKernelCUDA<<<
                            blocks, threads, 0, core::cuda::GetStream()>>>(
                            source_points.GetDataPtr<scalar_t>(),
                            source_colors.GetDataPtr<scalar_t>(),
                            target_points.GetDataPtr<scalar_t>(),
                            target_normals.GetDataPtr<scalar_t>(),
                            target_colors.GetDataPtr<scalar_t>(),
                            target_color_gradients.GetDataPtr<scalar_t>(),
                            correspondence_indices.GetDataPtr<int64_t>(),
                            sqrt_lambda_geometric, sqrt_lambda_photometric, n,
                            global_sum.GetDataPtr<scalar_t>(),
                            GetWeightFromRobustKernel);
                });
    });

    core::cuda::Synchronize();

    DecodeAndSolve6x6(global_sum, pose, residual, inlier_count);
}

template <typename scalar_t>
__global__ void ComputeInformationMatrixKernelCUDA(
        const scalar_t *target_points_ptr,
        const int64_t *correspondence_indices,
        const int n,
        scalar_t *global_sum) {
    __shared__ scalar_t local_sum0[kThread1DUnit];
    __shared__ scalar_t local_sum1[kThread1DUnit];
    __shared__ scalar_t local_sum2[kThread1DUnit];

    const int tid = threadIdx.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (workload_idx >= n) return;

    scalar_t J_x[6] = {0}, J_y[6] = {0}, J_z[6] = {0}, reduction[21] = {0};

    bool valid = GetInformationJacobians<scalar_t>(
            workload_idx, target_points_ptr, correspondence_indices, J_x, J_y,
            J_z);

    if (valid) {
        int i = 0;
        for (int j = 0; j < 6; ++j) {
            for (int k = 0; k <= j; ++k) {
                reduction[i] +=
                        J_x[j] * J_x[k] + J_y[j] * J_y[k] + J_z[j] * J_z[k];
                ++i;
            }
        }
    }

    ReduceSum6x6InformationJacobian<scalar_t, kThread1DUnit>(
            tid, valid, reduction, local_sum0, local_sum1, local_sum2,
            global_sum);
}

void ComputeInformationMatrixCUDA(const core::Tensor &target_points,
                                  const core::Tensor &correspondence_indices,
                                  core::Tensor &information_matrix,
                                  const core::Dtype &dtype,
                                  const core::Device &device) {
    int n = correspondence_indices.GetLength();

    core::Tensor global_sum = core::Tensor::Zeros({21}, dtype, device);
    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t *global_sum_ptr = global_sum.GetDataPtr<scalar_t>();

        ComputeInformationMatrixKernelCUDA<<<blocks, threads, 0,
                                             core::cuda::GetStream()>>>(
                target_points.GetDataPtr<scalar_t>(),
                correspondence_indices.GetDataPtr<int64_t>(), n,
                global_sum_ptr);

        core::cuda::Synchronize();

        core::Tensor global_sum_cpu =
                global_sum.To(core::Device("CPU:0"), core::Float64);
        double *sum_ptr = global_sum_cpu.GetDataPtr<double>();

        // Information matrix is on CPU of type Float64.
        double *GTG_ptr = information_matrix.GetDataPtr<double>();

        int i = 0;
        for (int j = 0; j < 6; j++) {
            for (int k = 0; k <= j; k++) {
                GTG_ptr[j * 6 + k] = GTG_ptr[k * 6 + j] = sum_ptr[i];
                ++i;
            }
        }
    });
}

}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
