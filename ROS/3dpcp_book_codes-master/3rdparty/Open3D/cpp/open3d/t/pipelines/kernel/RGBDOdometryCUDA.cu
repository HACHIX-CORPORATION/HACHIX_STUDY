#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/Dispatch.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/Tensor.h"
#include "open3d/t/geometry/kernel/GeometryIndexer.h"
#include "open3d/t/geometry/kernel/GeometryMacros.h"
#include "open3d/t/pipelines/kernel/RGBDOdometryImpl.h"
#include "open3d/t/pipelines/kernel/RGBDOdometryJacobianImpl.h"
#include "open3d/t/pipelines/kernel/Reduction6x6Impl.cuh"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {
namespace odometry {

__global__ void ComputeOdometryResultPointToPlaneCUDAKernel(
        NDArrayIndexer source_vertex_indexer,
        NDArrayIndexer target_vertex_indexer,
        NDArrayIndexer target_normal_indexer,
        TransformIndexer ti,
        float* global_sum,
        int rows,
        int cols,
        const float depth_outlier_trunc,
        const float depth_huber_delta) {
    const int kBlockSize = 256;
    __shared__ float local_sum0[kBlockSize];
    __shared__ float local_sum1[kBlockSize];
    __shared__ float local_sum2[kBlockSize];

    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    if (y >= rows || x >= cols) return;

    float J[6] = {0}, reduction[21 + 6 + 2];
    float r = 0;
    bool valid = GetJacobianPointToPlane(
            x, y, depth_outlier_trunc, source_vertex_indexer,
            target_vertex_indexer, target_normal_indexer, ti, J, r);

    float d_huber = HuberDeriv(r, depth_huber_delta);
    float r_huber = HuberLoss(r, depth_huber_delta);

    // Dump J, r into JtJ and Jtr
    int offset = 0;
    for (int i = 0; i < 6; ++i) {
        for (int j = 0; j <= i; ++j) {
            reduction[offset++] = J[i] * J[j];
        }
    }
    for (int i = 0; i < 6; ++i) {
        reduction[offset++] = J[i] * d_huber;
    }
    reduction[offset++] = r_huber;
    reduction[offset++] = valid;

    // Sum reduction: JtJ(21) and Jtr(6)
    for (size_t i = 0; i < 27; i += 3) {
        local_sum0[tid] = valid ? reduction[i + 0] : 0;
        local_sum1[tid] = valid ? reduction[i + 1] : 0;
        local_sum2[tid] = valid ? reduction[i + 2] : 0;
        __syncthreads();

        BlockReduceSum<float, kBlockSize>(tid, local_sum0, local_sum1,
                                          local_sum2);

        if (tid == 0) {
            atomicAdd(&global_sum[i + 0], local_sum0[0]);
            atomicAdd(&global_sum[i + 1], local_sum1[0]);
            atomicAdd(&global_sum[i + 2], local_sum2[0]);
        }
        __syncthreads();
    }

    // Sum reduction: residual(1) and inlier(1)
    {
        local_sum0[tid] = valid ? reduction[27] : 0;
        local_sum1[tid] = valid ? reduction[28] : 0;
        __syncthreads();

        BlockReduceSum<float, kBlockSize>(tid, local_sum0, local_sum1);
        if (tid == 0) {
            atomicAdd(&global_sum[27], local_sum0[0]);
            atomicAdd(&global_sum[28], local_sum1[0]);
        }
        __syncthreads();
    }
}

void ComputeOdometryResultPointToPlaneCUDA(
        const core::Tensor& source_vertex_map,
        const core::Tensor& target_vertex_map,
        const core::Tensor& target_normal_map,
        const core::Tensor& intrinsics,
        const core::Tensor& init_source_to_target,
        core::Tensor& delta,
        float& inlier_residual,
        int& inlier_count,
        const float depth_outlier_trunc,
        const float depth_huber_delta) {
    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);
    NDArrayIndexer target_vertex_indexer(target_vertex_map, 2);
    NDArrayIndexer target_normal_indexer(target_normal_map, 2);

    core::Device device = source_vertex_map.GetDevice();

    core::Tensor trans = init_source_to_target;
    TransformIndexer ti(intrinsics, trans);

    const int64_t rows = source_vertex_indexer.GetShape(0);
    const int64_t cols = source_vertex_indexer.GetShape(1);

    core::Tensor global_sum = core::Tensor::Zeros({29}, core::Float32, device);
    float* global_sum_ptr = global_sum.GetDataPtr<float>();

    const int kThreadSize = 16;
    const dim3 blocks((cols + kThreadSize - 1) / kThreadSize,
                      (rows + kThreadSize - 1) / kThreadSize);
    const dim3 threads(kThreadSize, kThreadSize);
    ComputeOdometryResultPointToPlaneCUDAKernel<<<blocks, threads, 0,
                                                  core::cuda::GetStream()>>>(
            source_vertex_indexer, target_vertex_indexer, target_normal_indexer,
            ti, global_sum_ptr, rows, cols, depth_outlier_trunc,
            depth_huber_delta);
    core::cuda::Synchronize();
    DecodeAndSolve6x6(global_sum, delta, inlier_residual, inlier_count);
}

__global__ void ComputeOdometryResultIntensityCUDAKernel(
        NDArrayIndexer source_depth_indexer,
        NDArrayIndexer target_depth_indexer,
        NDArrayIndexer source_intensity_indexer,
        NDArrayIndexer target_intensity_indexer,
        NDArrayIndexer target_intensity_dx_indexer,
        NDArrayIndexer target_intensity_dy_indexer,
        NDArrayIndexer source_vertex_indexer,
        TransformIndexer ti,
        float* global_sum,
        int rows,
        int cols,
        const float depth_outlier_trunc,
        const float intensity_huber_delta) {
    const int kBlockSize = 256;
    __shared__ float local_sum0[kBlockSize];
    __shared__ float local_sum1[kBlockSize];
    __shared__ float local_sum2[kBlockSize];

    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    if (y >= rows || x >= cols) return;

    float J[6] = {0}, reduction[21 + 6 + 2];
    float r = 0;
    bool valid = GetJacobianIntensity(
            x, y, depth_outlier_trunc, source_depth_indexer,
            target_depth_indexer, source_intensity_indexer,
            target_intensity_indexer, target_intensity_dx_indexer,
            target_intensity_dy_indexer, source_vertex_indexer, ti, J, r);

    float d_huber = HuberDeriv(r, intensity_huber_delta);
    float r_huber = HuberLoss(r, intensity_huber_delta);

    // Dump J, r into JtJ and Jtr
    int offset = 0;
    for (int i = 0; i < 6; ++i) {
        for (int j = 0; j <= i; ++j) {
            reduction[offset++] = J[i] * J[j];
        }
    }
    for (int i = 0; i < 6; ++i) {
        reduction[offset++] = J[i] * HuberDeriv(r, intensity_huber_delta);
    }
    reduction[offset++] = HuberLoss(r, intensity_huber_delta);
    reduction[offset++] = valid;

    ReduceSum6x6LinearSystem<float, kBlockSize>(tid, valid, reduction,
                                                local_sum0, local_sum1,
                                                local_sum2, global_sum);
}

void ComputeOdometryResultIntensityCUDA(
        const core::Tensor& source_depth,
        const core::Tensor& target_depth,
        const core::Tensor& source_intensity,
        const core::Tensor& target_intensity,
        const core::Tensor& target_intensity_dx,
        const core::Tensor& target_intensity_dy,
        const core::Tensor& source_vertex_map,
        const core::Tensor& intrinsics,
        const core::Tensor& init_source_to_target,
        core::Tensor& delta,
        float& inlier_residual,
        int& inlier_count,
        const float depth_outlier_trunc,
        const float intensity_huber_delta) {
    NDArrayIndexer source_depth_indexer(source_depth, 2);
    NDArrayIndexer target_depth_indexer(target_depth, 2);

    NDArrayIndexer source_intensity_indexer(source_intensity, 2);
    NDArrayIndexer target_intensity_indexer(target_intensity, 2);

    NDArrayIndexer target_intensity_dx_indexer(target_intensity_dx, 2);
    NDArrayIndexer target_intensity_dy_indexer(target_intensity_dy, 2);

    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);

    core::Device device = source_vertex_map.GetDevice();
    core::Tensor trans = init_source_to_target;
    t::geometry::kernel::TransformIndexer ti(intrinsics, trans);

    const int64_t rows = source_vertex_indexer.GetShape(0);
    const int64_t cols = source_vertex_indexer.GetShape(1);

    core::Tensor global_sum = core::Tensor::Zeros({29}, core::Float32, device);
    float* global_sum_ptr = global_sum.GetDataPtr<float>();

    const int kThreadSize = 16;
    const dim3 blocks((cols + kThreadSize - 1) / kThreadSize,
                      (rows + kThreadSize - 1) / kThreadSize);
    const dim3 threads(kThreadSize, kThreadSize);
    ComputeOdometryResultIntensityCUDAKernel<<<blocks, threads, 0,
                                               core::cuda::GetStream()>>>(
            source_depth_indexer, target_depth_indexer,
            source_intensity_indexer, target_intensity_indexer,
            target_intensity_dx_indexer, target_intensity_dy_indexer,
            source_vertex_indexer, ti, global_sum_ptr, rows, cols,
            depth_outlier_trunc, intensity_huber_delta);
    core::cuda::Synchronize();
    DecodeAndSolve6x6(global_sum, delta, inlier_residual, inlier_count);
}

__global__ void ComputeOdometryResultHybridCUDAKernel(
        NDArrayIndexer source_depth_indexer,
        NDArrayIndexer target_depth_indexer,
        NDArrayIndexer source_intensity_indexer,
        NDArrayIndexer target_intensity_indexer,
        NDArrayIndexer target_depth_dx_indexer,
        NDArrayIndexer target_depth_dy_indexer,
        NDArrayIndexer target_intensity_dx_indexer,
        NDArrayIndexer target_intensity_dy_indexer,
        NDArrayIndexer source_vertex_indexer,
        TransformIndexer ti,
        float* global_sum,
        int rows,
        int cols,
        const float depth_outlier_trunc,
        const float depth_huber_delta,
        const float intensity_huber_delta) {
    const int kBlockSize = 256;
    __shared__ float local_sum0[kBlockSize];
    __shared__ float local_sum1[kBlockSize];
    __shared__ float local_sum2[kBlockSize];

    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    if (y >= rows || x >= cols) return;

    float J_I[6] = {0}, J_D[6] = {0}, reduction[21 + 6 + 2];
    float r_I = 0, r_D = 0;
    bool valid = GetJacobianHybrid(
            x, y, depth_outlier_trunc, source_depth_indexer,
            target_depth_indexer, source_intensity_indexer,
            target_intensity_indexer, target_depth_dx_indexer,
            target_depth_dy_indexer, target_intensity_dx_indexer,
            target_intensity_dy_indexer, source_vertex_indexer, ti, J_I, J_D,
            r_I, r_D);

    float d_huber_D = HuberDeriv(r_D, depth_huber_delta);
    float d_huber_I = HuberDeriv(r_I, intensity_huber_delta);

    float r_huber_D = HuberLoss(r_D, depth_huber_delta);
    float r_huber_I = HuberLoss(r_I, intensity_huber_delta);

    // Dump J, r into JtJ and Jtr
    int offset = 0;
    for (int i = 0; i < 6; ++i) {
        for (int j = 0; j <= i; ++j) {
            reduction[offset++] = J_I[i] * J_I[j] + J_D[i] * J_D[j];
        }
    }
    for (int i = 0; i < 6; ++i) {
        reduction[offset++] = J_I[i] * d_huber_I + J_D[i] * d_huber_D;
    }
    reduction[offset++] = r_huber_D + r_huber_I;
    reduction[offset++] = valid;

    ReduceSum6x6LinearSystem<float, kBlockSize>(tid, valid, reduction,
                                                local_sum0, local_sum1,
                                                local_sum2, global_sum);
}

void ComputeOdometryResultHybridCUDA(const core::Tensor& source_depth,
                                     const core::Tensor& target_depth,
                                     const core::Tensor& source_intensity,
                                     const core::Tensor& target_intensity,
                                     const core::Tensor& target_depth_dx,
                                     const core::Tensor& target_depth_dy,
                                     const core::Tensor& target_intensity_dx,
                                     const core::Tensor& target_intensity_dy,
                                     const core::Tensor& source_vertex_map,
                                     const core::Tensor& intrinsics,
                                     const core::Tensor& init_source_to_target,
                                     core::Tensor& delta,
                                     float& inlier_residual,
                                     int& inlier_count,
                                     const float depth_outlier_trunc,
                                     const float depth_huber_delta,
                                     const float intensity_huber_delta) {
    NDArrayIndexer source_depth_indexer(source_depth, 2);
    NDArrayIndexer target_depth_indexer(target_depth, 2);

    NDArrayIndexer source_intensity_indexer(source_intensity, 2);
    NDArrayIndexer target_intensity_indexer(target_intensity, 2);

    NDArrayIndexer target_depth_dx_indexer(target_depth_dx, 2);
    NDArrayIndexer target_depth_dy_indexer(target_depth_dy, 2);
    NDArrayIndexer target_intensity_dx_indexer(target_intensity_dx, 2);
    NDArrayIndexer target_intensity_dy_indexer(target_intensity_dy, 2);

    NDArrayIndexer source_vertex_indexer(source_vertex_map, 2);

    core::Device device = source_vertex_map.GetDevice();
    core::Tensor trans = init_source_to_target;
    t::geometry::kernel::TransformIndexer ti(intrinsics, trans);

    const int64_t rows = source_vertex_indexer.GetShape(0);
    const int64_t cols = source_vertex_indexer.GetShape(1);

    core::Tensor global_sum = core::Tensor::Zeros({29}, core::Float32, device);
    float* global_sum_ptr = global_sum.GetDataPtr<float>();

    const int kThreadSize = 16;
    const dim3 blocks((cols + kThreadSize - 1) / kThreadSize,
                      (rows + kThreadSize - 1) / kThreadSize);
    const dim3 threads(kThreadSize, kThreadSize);
    ComputeOdometryResultHybridCUDAKernel<<<blocks, threads, 0,
                                            core::cuda::GetStream()>>>(
            source_depth_indexer, target_depth_indexer,
            source_intensity_indexer, target_intensity_indexer,
            target_depth_dx_indexer, target_depth_dy_indexer,
            target_intensity_dx_indexer, target_intensity_dy_indexer,
            source_vertex_indexer, ti, global_sum_ptr, rows, cols,
            depth_outlier_trunc, depth_huber_delta, intensity_huber_delta);
    core::cuda::Synchronize();
    DecodeAndSolve6x6(global_sum, delta, inlier_residual, inlier_count);
}

}  // namespace odometry
}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
